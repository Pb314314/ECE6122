/*
Author: Bo Pang
Class:  ECE6122  2023-11-6
Description: This contains the implementation of Lab4.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cmath>
#include <chrono>

#define NUM_BLOCKS 256
#define THREADS_PER_BLOCK 256

#define RADIUS 1.0
using namespace std;
__global__ void calculate_distance(float* results, unsigned int seed, int step_num) 
{
    int tid = threadIdx.x;
    hiprandState state;
    hiprand_init(seed, tid, 0, &state);

    int x = 0;
    int y = 0;
    for (int i = 0; i < step_num; ++i) {
        float random = hiprand_uniform(&state);
        if(random < 0.25) x++;
        else if(random < 0.5) x--;
        else if(random < 0.75) y++;
        else if(random < 1.00) y--;
    }

    results[tid] = x*x + y*y;
}

void cudaMalloc_random_walk(int user_num, int step_num){
    auto start_time  = std::chrono::high_resolution_clock::now();

    float* d_results;
    float* h_results = new float[user_num];

    hipMalloc((void**)&d_results, sizeof(float) * user_num);

    calculate_distance<<<1, user_num>>>(d_results, time(NULL), step_num); // every thread simulate one random walk
    hipMemcpy(h_results, d_results, sizeof(float) * user_num, hipMemcpyDeviceToHost);

    float distance = 0;
    for (int i = 0; i < user_num; ++i) {
        //cout<< sqrt(h_results[i]) <<endl;
        distance += sqrt(h_results[i]);
    }
    distance /= user_num;

    
    hipFree(d_results);
    delete[] h_results;

    auto end_time = std::chrono::high_resolution_clock::now();

    auto duration_us = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    std::cout << "    Time to calculate:(microseconds): " << duration_us.count() << std::endl;
    std::cout << "    Average distance from origin: " << distance << std::endl;
    return;
}

void cudaMallocHost_random_walk(int user_num, int step_num){
    auto start_time  = std::chrono::high_resolution_clock::now();

    float* d_results;
    float* h_results;
    hipHostMalloc(&h_results, sizeof(float) * user_num, hipHostMallocDefault);
    hipHostMalloc(&d_results, sizeof(float) * user_num, hipHostMallocDefault);

    calculate_distance<<<1, user_num>>>(d_results, time(NULL), step_num); // every thread simulate one random walk
    hipMemcpy(h_results, d_results, sizeof(float) * user_num, hipMemcpyDeviceToHost);

    float distance = 0;
    for (int i = 0; i < user_num; ++i) {
        //cout<< sqrt(h_results[i]) <<endl;
        distance += sqrt(h_results[i]);
    }
    distance /= user_num;

    
    hipHostFree(d_results);
    hipHostFree(h_results);

    auto end_time = std::chrono::high_resolution_clock::now();

    auto duration_us = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    std::cout << "    Time to calculate:(microseconds): " << duration_us.count() << std::endl;
    std::cout << "    Average distance from origin: " << distance << std::endl;
    return;
}


void cudaMallocManaged_random_walk(int user_num, int step_num){
    auto start_time  = std::chrono::high_resolution_clock::now();

    float* d_results;
    float* h_results;
    hipMallocManaged(&h_results, sizeof(float) * user_num);
    hipMallocManaged(&d_results, sizeof(float) * user_num);

    calculate_distance<<<1, user_num>>>(d_results, time(NULL), step_num); // every thread simulate one random walk
    hipMemcpy(h_results, d_results, sizeof(float) * user_num, hipMemcpyDeviceToHost);

    float distance = 0;
    for (int i = 0; i < user_num; ++i) {
        //cout<< sqrt(h_results[i]) <<endl;
        distance += sqrt(h_results[i]);
    }
    distance /= user_num;

    
    hipFree(d_results);
    hipFree(h_results);

    auto end_time = std::chrono::high_resolution_clock::now();

    auto duration_us = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    std::cout << "    Time to calculate:(microseconds): " << duration_us.count() << std::endl;
    std::cout << "    Average distance from origin: " << distance << std::endl;
    return;
}

int main(int argc, char** argv) 
{
    int user_num = 1000;
    int step_num = 1000;
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-W") == 0 && i + 1 < argc) {
            user_num = atoi(argv[i + 1]);
        } else if (strcmp(argv[i], "-I") == 0 && i + 1 < argc) {
            step_num = atoi(argv[i + 1]);
        }
    }
    //cout<< "Users: "<< user_num <<" Steps: "<< step_num <<endl;

    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        return 1;
    }

    int integratedDevice = -1; // Initialize with an invalid device index

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, device);

        if (!deviceProps.integrated) {
            integratedDevice = device;
            break; // Found an not integrated device
        }
    }
    if (integratedDevice == -1) {
        std::cerr << "No integrated GPU devices found." << std::endl;
        return 1;
    }
    hipSetDevice(integratedDevice);

    cout<<"Normal CUDA memory Allocation:"<<endl;
    cudaMalloc_random_walk(user_num,step_num);

    cout<<"Pinned CUDA memory Allocation:"<<endl;
    cudaMallocHost_random_walk(user_num,step_num);
    
    cout<< "Managed CUDA memory Allocation:"<<endl;
    cudaMallocManaged_random_walk(user_num,step_num);
    return 0;
}
